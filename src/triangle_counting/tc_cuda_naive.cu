#include "hip/hip_runtime.h"
#include "tc.h"

__global__ void kernel_tc_naive(
		uint64_t* __restrict__ sum,
		const uint32_t* __restrict__ index,
		const uint32_t* __restrict__ dest,
		const uint32_t N ){

	const uint32_t u = blockIdx.x*blockDim.y + threadIdx.y ;
	if( u >= N )
		return ;

	const uint32_t start_u = index[u] ;
	const uint32_t end_u = index[u+1] ;

	uint32_t val = 0 ;
	for( uint32_t i=index[u]+threadIdx.x ; i<end_u ; i+=blockDim.x ){
		const uint32_t v = dest[i] ;

		// set index
		uint32_t end_v = index[v+1] ;
		uint32_t iv = index[v] ;
		if( iv >= end_v )
			continue ;

		uint32_t iu = start_u ;

		// fetch dest
		uint32_t du = dest[iu] ;
		uint32_t dv = dest[iv] ;

		for(;;){
			// update dest[] of u
			if( du <= dv ){
				// equal
				if( du == dv )
					val ++ ;

				if( ++iu >= end_u )
					break ;
				du = dest[iu] ;
			}

			// update dest[] of v
			else{
				if( ++iv >= end_v )
					break ;
				dv = dest[iv] ;
			}
		}
	}

	// reduce
	#pragma unroll
	for ( int offset = warpSize/2; offset > 0; offset /= 2 )
		val += __shfl_down( val, offset );
	if( threadIdx.x == 0 && val > 0 )
		atomicAdd( (unsigned long long*)sum, (unsigned long long)val ) ;
}

uint64_t triangle_counting_cuda_naive( const Edge_Array_Device &edge ){
	const uint32_t N = (uint32_t)edge.get_N() ;
	const uint32_t *index = edge.get_index() ;
	const uint32_t *dest = edge.get_dest() ;

	uint64_t *sum ;
	hipMallocManaged( &sum, sizeof(uint64_t) ) ;
	*sum = 0 ;

	dim3 threads(32,32) ;
	dim3 blocks((N+threads.y-1)/threads.y) ;
	kernel_tc_naive <<< blocks, threads >>> ( sum, index, dest, N ) ;
	hipDeviceSynchronize() ;

	uint64_t re = *sum ;
	hipFree( sum ) ;

	return re ;
}

