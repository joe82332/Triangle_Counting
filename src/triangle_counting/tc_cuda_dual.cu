#include "hip/hip_runtime.h"
#include "tc.h"

__global__ void kernel_tc_dual(
		uint64_t* __restrict__ sum,
		const uint32_t* __restrict__ index,
		const uint32_t* __restrict__ dest,
		const uint32_t N ){

	const uint32_t u = blockIdx.x*blockDim.z + threadIdx.z ;
	if( u >= N )
		return ;

	const uint32_t end_u = index[u+1] ;

	uint32_t val = 0 ;
	for( uint32_t i=index[u]+threadIdx.x ; i<end_u ; i+=blockDim.x ){
		uint32_t v ;
		if( threadIdx.y == 0 )
			v = u ;
		else
			v = dest[i] ;

		// set index
		uint32_t iv = index[v] ;
		const uint32_t end_v = index[v+1] ;

		// fetch dest
		uint32_t dv ;

		int done = 0 ;
		if( iv >= end_v )
			done = 1 ;
		else
			dv = dest[iv] ;

		for(;;){
			done += __shfl_xor( done, 16 ) ;
			if( done )
				break ;

			uint32_t du = __shfl_xor( dv, 16 ) ;

			// update dest[] of v
			if( dv <= du ){
				// equal
				if( dv == du )
					val ++ ;

				if( ++iv >= end_v )
					done = 1 ;
				else
					dv = dest[iv] ;
			}
		}
	}

	// reduce
	#pragma unroll
	for ( int offset = blockDim.x/2; offset > 0; offset /= 2 )
		val += __shfl_down( val, offset );
	if( threadIdx.y + threadIdx.x == 0 && val > 0 )
		atomicAdd( (unsigned long long*)sum, (unsigned long long)val ) ;
}


uint64_t triangle_counting_cuda_dual( const Edge_Array_Device &edge ){
	const uint32_t N = (uint32_t)edge.get_N() ;
	const uint32_t *index = edge.get_index() ;
	const uint32_t *dest = edge.get_dest() ;

	uint64_t *sum ;
	hipMallocManaged( &sum, sizeof(uint64_t) ) ;
	*sum = 0 ;

	dim3 threads(16,2,32) ;
	dim3 blocks((N+threads.z-1)/threads.z) ;
	kernel_tc_dual <<< blocks, threads >>> ( sum, index, dest, N ) ;
	hipDeviceSynchronize() ;

	uint64_t re = *sum ;
	hipFree( sum ) ;

	return re ;
}

