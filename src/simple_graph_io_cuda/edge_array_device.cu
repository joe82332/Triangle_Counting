#include "edge_io_cuda.h"

// index[] points to the begin of dest[]
Edge_Array_Device::Edge_Array_Device( const Edge_List &b ) : Edge_Array(b) {
	uint32_t *_index = index ;
	uint32_t *_dest = dest ;

	hipMalloc( &index, (N+1)*sizeof(uint32_t) ) ;	// [0] ~ [N]
	hipMalloc( &dest,  (E+1)*sizeof(uint32_t) ) ;	// +1 avoids empty allocation

	hipMemcpy( index, _index, (N+1)*sizeof(uint32_t), hipMemcpyDefault ) ;
	hipMemcpy( dest, _dest, (E+1)*sizeof(uint32_t), hipMemcpyDefault ) ;

	delete [] _index ;
	delete [] _dest ;
}
Edge_Array_Device::~Edge_Array_Device(){
	if( index ){
		hipFree( index ) ;
		index = NULL ;
	}

	if( dest ){
		hipFree( dest ) ;
		dest = NULL ;
	}
}


