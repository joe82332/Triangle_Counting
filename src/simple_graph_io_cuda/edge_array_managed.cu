#include "edge_io_cuda.h"

// index[] points to the begin of dest[]
Edge_Array_Managed::Edge_Array_Managed( const Edge_List &b ){
	N = b.get_N() ;
	E = b.get_E() ;

	hipMallocManaged( &index, (N+1)*sizeof(uint32_t) ) ;	// [0] ~ [N]
	hipMallocManaged( &dest,  (E+1)*sizeof(uint32_t) ) ;	// +1 avoids empty allocation

	fill( b ) ;
}
Edge_Array_Managed::~Edge_Array_Managed(){
	if( index ){
		hipFree( index ) ;
		index = NULL ;
	}

	if( dest ){
		hipFree( dest ) ;
		dest = NULL ;
	}
}


